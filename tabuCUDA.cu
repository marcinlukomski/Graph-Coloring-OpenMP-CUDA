#include <iostream>
#include <vector>
#include <set>
#include <map>
#include <algorithm>
#include <cstdlib>
#include <ctime>
#include <chrono>
#include <fstream>
#include <hip/hip_runtime.h>

__global__ void count_conflicts(int* d_current_colors, int* d_adjacency_list, int* d_offsets, int num_nodes, int* d_conflict_count) {
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    if (idx < num_nodes) {
        int color = d_current_colors[idx];
        int start = d_offsets[idx];
        int end = d_offsets[idx + 1];
        for (int i = start; i < end; ++i) {
            if (color == d_current_colors[d_adjacency_list[i]]) {
                atomicAdd(d_conflict_count, 1);
            }
        }
    }
}

void tabu_search_cuda(int num_nodes, int max_colors, const std::vector<std::vector<int>>& adjacency_list, bool debug = false) {
    srand(time(NULL));

    int max_iterations = 5000;    
    int tabu_list_size = 4;       
    int neighbor_reps = 700;      

    int conflict_count, new_conflict_count;
    int selected_candidate;
    int solutions_checked = 0;

    std::vector<int> tabu_list;
    std::set<int> candidate_set;
    std::vector<int> candidates;
    std::vector<int> current_colors(num_nodes);
    std::vector<int> candidate_colors(num_nodes);
    std::map<int, int> aspiration_criteria;
    std::vector<std::pair<int, int>> debug_info;

    if (max_colors > num_nodes) max_colors = num_nodes;

    for (int i = 0; i < num_nodes; i++) {
        current_colors[i] = rand() % max_colors;
    }

    std::cout << "Initial coloring:";
    for (int i = 0; i < num_nodes; i++) {
        std::cout << " " << current_colors[i];
    }
    std::cout << std::endl;

    int* d_current_colors;
    int* d_adjacency_list;
    int* d_offsets;
    int* d_conflict_count;
    int* d_candidate_colors;

    int adjacency_list_size = 0;
    for (const auto& neighbors : adjacency_list) {
        adjacency_list_size += neighbors.size();
    }

    std::vector<int> flat_adjacency_list(adjacency_list_size);
    std::vector<int> offsets(num_nodes + 1);

    int index = 0;
    for (int i = 0; i < num_nodes; ++i) {
        offsets[i] = index;
        for (int neighbor : adjacency_list[i]) {
            flat_adjacency_list[index++] = neighbor;
        }
    }
    offsets[num_nodes] = index;

    hipMalloc(&d_current_colors, num_nodes * sizeof(int));
    hipMalloc(&d_adjacency_list, adjacency_list_size * sizeof(int));
    hipMalloc(&d_offsets, (num_nodes + 1) * sizeof(int));
    hipMalloc(&d_conflict_count, sizeof(int));
    hipMalloc(&d_candidate_colors, num_nodes * sizeof(int));

    hipMemcpy(d_current_colors, current_colors.data(), num_nodes * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_adjacency_list, flat_adjacency_list.data(), adjacency_list_size * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_offsets, offsets.data(), (num_nodes + 1) * sizeof(int), hipMemcpyHostToDevice);

    dim3 blockSize(256);
    dim3 gridSize((num_nodes + blockSize.x - 1) / blockSize.x);

    while (max_iterations > 0) {
        candidate_set.clear();
        conflict_count = 0;

        int h_conflict_count = 0;
        hipMemcpy(d_conflict_count, &h_conflict_count, sizeof(int), hipMemcpyHostToDevice);

        count_conflicts<<<gridSize, blockSize>>>(d_current_colors, d_adjacency_list, d_offsets, num_nodes, d_conflict_count);
        hipMemcpy(&conflict_count, d_conflict_count, sizeof(int), hipMemcpyDeviceToHost);
        conflict_count /= 2;

        if (conflict_count == 0) break;
        solutions_checked++;

        candidates.clear();
        for (int i = 0; i < num_nodes; ++i) {
            if (std::find(tabu_list.begin(), tabu_list.end(), i) == tabu_list.end()) {
                for (int neighbor : adjacency_list[i]) {
                    if (current_colors[i] == current_colors[neighbor]) {
                        candidate_set.insert(i);
                        break;
                    }
                }
            }
        }
        std::copy(candidate_set.begin(), candidate_set.end(), std::back_inserter(candidates));

        for (int i = 0; i < neighbor_reps; i++) {
            selected_candidate = candidates[rand() % candidates.size()];
            candidate_colors = current_colors;
            candidate_colors[selected_candidate] = rand() % (max_colors - 1);

            if (candidate_colors[selected_candidate] == current_colors[selected_candidate]) {
                candidate_colors[selected_candidate] = max_colors - 1;
            }

            hipMemcpy(d_candidate_colors, candidate_colors.data(), num_nodes * sizeof(int), hipMemcpyHostToDevice);
            h_conflict_count = 0;
            hipMemcpy(d_conflict_count, &h_conflict_count, sizeof(int), hipMemcpyHostToDevice);

            count_conflicts<<<gridSize, blockSize>>>(d_candidate_colors, d_adjacency_list, d_offsets, num_nodes, d_conflict_count);
            hipMemcpy(&new_conflict_count, d_conflict_count, sizeof(int), hipMemcpyDeviceToHost);
            new_conflict_count /= 2;

            if (new_conflict_count < conflict_count) {
                bool in_tabu_list = false;

                if (aspiration_criteria.find(conflict_count) == aspiration_criteria.end()) {
                    aspiration_criteria.insert({conflict_count, conflict_count - 1});
                }

                if (new_conflict_count <= aspiration_criteria[conflict_count]) {
                    aspiration_criteria[conflict_count] = new_conflict_count - 1;

                    for (auto x : tabu_list) {
                        if (x == selected_candidate) {
                            tabu_list.erase(std::remove(tabu_list.begin(), tabu_list.end(), selected_candidate), tabu_list.end());
                            break;
                        }
                    }
                    break;
                } else {
                    for (auto x : tabu_list) {
                        if (x == selected_candidate) {
                            in_tabu_list = true;
                            break;
                        }
                    }
                    if (in_tabu_list) {
                        continue;
                    } else {
                        break;
                    }
                }
            }
        }

        tabu_list.push_back(selected_candidate);
        if (tabu_list.size() > tabu_list_size) {
            tabu_list.erase(tabu_list.begin());
        }
        current_colors = candidate_colors;

        hipMemcpy(d_current_colors, current_colors.data(), num_nodes * sizeof(int), hipMemcpyHostToDevice);

        max_iterations--;
    }

    hipFree(d_current_colors);
    hipFree(d_adjacency_list);
    hipFree(d_offsets);
    hipFree(d_conflict_count);
    hipFree(d_candidate_colors);

    std::cout << "Final coloring:";
    for (int i = 0; i < num_nodes; i++) {
        std::cout << " " << current_colors[i];
    }
    std::cout << std::endl;

    int total_conflicts = 0;
    for (int i = 0; i < num_nodes; i++) {
        for (auto neighbor : adjacency_list[i]) {
            if (current_colors[i] == current_colors[neighbor]) {
                debug_info.push_back(std::make_pair(neighbor, i));
                if (std::find(debug_info.begin(), debug_info.end(), std::make_pair(i, neighbor)) == debug_info.end()) {
                    std::cout << "Conflict: " << i << " " << neighbor << " - color " << current_colors[i] << std::endl;
                }
                total_conflicts++;
            }
        }
    }
    total_conflicts /= 2;
    std::cout << "Number of conflicts: " << total_conflicts << std::endl;
}

std::vector<std::vector<int>> read_graph_from_file(const std::string& filename, int& num_nodes) {
    std::ifstream infile(filename);
    if (!infile) {
        std::cerr << "Error opening file." << std::endl;
        exit(1);
    }

    infile >> num_nodes;
    std::vector<std::vector<int>> adjacency_list(num_nodes);

    int u, v;
    while (infile >> u >> v) {
        adjacency_list[u - 1].push_back(v - 1);
        adjacency_list[v - 1].push_back(u - 1);
    }

    return adjacency_list;
}

int main() {
    std::string filename = "graph.txt";
    int num_nodes;
    std::vector<std::vector<int>> adjacency_list = read_graph_from_file(filename, num_nodes);

    int max_colors = 3;

    auto start = std::chrono::high_resolution_clock::now();
    tabu_search_cuda(num_nodes, max_colors, adjacency_list);
    auto end = std::chrono::high_resolution_clock::now();

    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time: " << duration.count() << " seconds" << std::endl;

    return 0;
}
